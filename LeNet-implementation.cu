
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>

void MatrixInit(float *M, int n, int p, int d, float coeff){ //n,p,d
	int i,j,k;
	if (d==0){
		for (i = 0; i < n; i++){
			for (j = 0; j < p; j++){
				
				M[i*p+j] = (float)(rand()*coeff)/RAND_MAX;
			}
		}
	}
	else{
		for (i = 0; i < n; i++){
			for (j = 0; j < p; j++){
				for (k = 0; k < d; k++){
					M[i*p*d+j*d+k] = (float)(rand()*coeff)/RAND_MAX;
				}
			}
		}
	}
}

void MatrixPrint2D(float *M, int n, int p){
	for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
            printf("%.2f\t", M[i * p + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixPrint3D(float *M, int n, int p, int d){
	for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
			for (int k = 0; k < d; k++){
				printf("%.2f\t", M[i * p * d + j * d + k]);
			}
			printf("\n");
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
	int i,j;
	for (i = 0; i < n; i++){
		for (j = 0; j < p; j++){
			Mout[i*p+j] = M1[i*p+j] + M2[i*p+j] ;
		}
	}
}

__global__ void cudaMatrixAdd (float *M1, float *M2, float *M3) {
	M3[blockIdx.x] = M1[blockIdx.x] + M2[blockIdx.x];
}

void MatrixMult(float *M1, float *M2, float *Mout, int n){
	int i,j;
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			float product = 0;
			for (int k = 0; k<n; k++){
				product += M1[i*n+k]*M2[k*n+j];
			}
			Mout[i*n+j] = product;
		}
	}
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row < n && col < n){
		float product = 0;
		for (int k = 0; k<n; k++){
			product += M1[row*n+k]*M2[k*n+col];
		}
		Mout[row*n+col] = product;
	}
}

__global__ void cudaMatrixConv2D(float *data, float *kernel, float *Mout, int n_data, int p_data, int n_kernel, int p_kernel){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	float coeff;
		
	if ( (n_kernel/2)<=row && row<(n_data-n_kernel/2) && (p_kernel/2)<=col && col<(p_data-p_kernel/2) ){
		float result = 0;
		for (int i = 0; i<n_kernel; i++){
			for(int j = 0; j<p_kernel; j++){
				
				coeff = data[(row+i-n_kernel/2)*n_data + col+j-p_kernel/2]*kernel[i*n_kernel+j];
				result += coeff;
			}
		}	
		Mout[(row-n_kernel/2)*(n_data-n_kernel+1)+col-p_kernel/2] = result;
	}
}


__global__ void cudaMatrixConv3D(float *data, float *kernel, float *Mout, int n_data, int p_data, int n_kernel, int p_kernel, int d_kernel){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	float coeff;
		
	if ( (n_kernel/2)<=row && row<(n_data-n_kernel/2) && (p_kernel/2)<=col && col<(p_data-p_kernel/2) ){
		for (int k = 0; k<d_kernel; k++){
			float result = 0;
			for (int i = 0; i<n_kernel; i++){
				for(int j = 0; j<p_kernel; j++){
					coeff = data[(row+i-n_kernel/2)*n_data + col+j-p_kernel/2]*kernel[k*n_kernel*p_kernel+i*n_kernel+j];
					result += coeff;
				}
			}
			Mout[k*(n_data-n_kernel/2-1)*(p_data-p_kernel/2-1)+(row-n_kernel/2)*(n_data-n_kernel+1)+col-p_kernel/2] = result;
		}
	}
}

__global__ void cudaMatrixSubSampling3D(float *data, float *Mout, int n_data, int p_data, int d_data){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row>=0 && row<n_data/2 && col>=0 && col<p_data/2){
		for (int k = 0; k<d_data; k++){
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + row*2*n_data + col*2]/4.0;
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + row*2*n_data + col*2 + 1]/4.0;
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + (row*2+1)*n_data + col*2]/4.0;
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + (row*2+1)*n_data + col*2+1]/4.0;
		}
	}
}


int main(void){
	srand(time(NULL));
	int raw_data_size = 32*32*sizeof(float);
	int C1_data_size = 6*28*28*sizeof(float);
	int S1_data_size = 6*14*14*sizeof(float);
	int C1_kernel_size = 6*5*5*sizeof(float);
	
	
	float *raw_data = (float*) malloc(raw_data_size);
	float *C1_data = (float*) malloc(C1_data_size);
	float *S1_data = (float*) malloc(S1_data_size);
	float *C1_kernel = (float*) malloc(C1_kernel_size);
	
	MatrixInit(raw_data, 32, 32, 0, 1);
	MatrixInit(C1_data, 6, 28, 28, 0);
	MatrixInit(S1_data, 6, 14, 14, 0);
	MatrixInit(C1_kernel, 6, 5, 5, 1);
	
	
	int n_kernel = 3;
	int n_data = 6;
	int d_kernel = 2;
	float *test_data = (float*) malloc(n_data*n_data*sizeof(float));
	float *test_kernel = (float*) malloc(n_kernel*n_kernel*d_kernel*sizeof(float));
	MatrixInit(test_data, n_data, n_data, 0, 1);
	MatrixInit(test_kernel, n_kernel, n_kernel, d_kernel, 1);
	
	
	float *test_conv = (float*) malloc((n_data-n_kernel+1)*(n_data-n_kernel+1)*d_kernel*sizeof(float));
	float *test_sub_samp = (float*) malloc(2*2*2*sizeof(float));
	
	int N = n_data*n_data;
	dim3 dimBlock(N,N);
	dim3 dimGrid(ceil(N/16.0), ceil(N/16.0));
	float *data_cu, *kernel_cu, *conv_cu, *sub_samp;
	hipMalloc((void **) &data_cu, n_data*n_data*sizeof(float));
	hipMalloc((void **) &kernel_cu, n_kernel*n_kernel*d_kernel*sizeof(float));
	hipMalloc((void **) &conv_cu, (n_data-n_kernel+1)*(n_data-n_kernel+1)*d_kernel*sizeof(float));
	hipMalloc((void **) &sub_samp, 2*2*2*sizeof(float));
	
	hipMemcpy(data_cu, test_data, n_data*n_data*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(kernel_cu, test_kernel, n_kernel*n_kernel*d_kernel*sizeof(float), hipMemcpyHostToDevice);
	cudaMatrixConv3D<<<dimBlock, dimGrid>>>(data_cu, kernel_cu, conv_cu, n_data, n_data, n_kernel, n_kernel, d_kernel);
	cudaMatrixSubSampling3D<<<dimBlock, dimGrid>>>(conv_cu, sub_samp, 4, 4, 2);
	hipMemcpy(test_conv, conv_cu, (n_data-n_kernel+1)*(n_data-n_kernel+1)*d_kernel*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(test_sub_samp, sub_samp, 2*2*2*sizeof(float), hipMemcpyDeviceToHost);
	
	printf("data : \n");
	MatrixPrint2D(test_data, n_data, n_data);
	printf("kernels : \n");
	MatrixPrint3D(test_kernel, d_kernel, n_kernel, n_kernel);
	printf("conv result : \n");
	MatrixPrint3D(test_conv, d_kernel, (n_data-n_kernel+1), (n_data-n_kernel+1));
	printf("sub samp result : \n");
	MatrixPrint3D(test_sub_samp, 2, 2, 2);
	
}
