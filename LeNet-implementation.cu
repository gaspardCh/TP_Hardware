
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>

void MatrixInit(float *M, int n, int p, int d, float coeff){ //n,p,d
	int i,j,k;
	if (d==0){
		for (i = 0; i < n; i++){
			for (j = 0; j < p; j++){
				
				M[i*p+j] = (float)(rand()*coeff)/RAND_MAX;
			}
		}
	}
	else{
		for (i = 0; i < n; i++){
			for (j = 0; j < p; j++){
				for (k = 0; k < d; k++){
					M[i*p*d+j*d+k] = (float)(rand()*coeff)/RAND_MAX;
				}
			}
		}
	}
}

void MatrixPrint2D(float *M, int n, int p){
	for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
            printf("%.2f\t", M[i * p + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixPrint3D(float *M, int n, int p, int d){
	for (int i = 0; i < n; ++i) {
        for (int j = 0; j < p; ++j) {
			for (int k = 0; k < d; k++){
				printf("%.2f\t", M[i * p * d + j * d + k]);
			}
			printf("\n");
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
	int i,j;
	for (i = 0; i < n; i++){
		for (j = 0; j < p; j++){
			Mout[i*p+j] = M1[i*p+j] + M2[i*p+j] ;
		}
	}
}

__global__ void cudaMatrixAdd (float *M1, float *M2, float *M3) {
	M3[blockIdx.x] = M1[blockIdx.x] + M2[blockIdx.x];
}

void MatrixMult(float *M1, float *M2, float *Mout, int n){
	int i,j;
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			float product = 0;
			for (int k = 0; k<n; k++){
				product += M1[i*n+k]*M2[k*n+j];
			}
			Mout[i*n+j] = product;
		}
	}
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row < n && col < n){
		float product = 0;
		for (int k = 0; k<n; k++){
			product += M1[row*n+k]*M2[k*n+col];
		}
		Mout[row*n+col] = product;
	}
}

__global__ void cudaMatrixConv2D(float *data, float *kernel, float *Mout, int n_data, int p_data, int n_kernel, int p_kernel){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	float coeff;
		
	if ( (n_kernel/2)<=row && row<(n_data-n_kernel/2) && (p_kernel/2)<=col && col<(p_data-p_kernel/2) ){
		float result = 0;
		for (int i = 0; i<n_kernel; i++){
			for(int j = 0; j<p_kernel; j++){
				
				coeff = data[(row+i-n_kernel/2)*n_data + col+j-p_kernel/2]*kernel[i*p_kernel+j];
				result += coeff;
			}
		}	
		Mout[(row-n_kernel/2)*(n_data-n_kernel+1)+col-p_kernel/2] = result;
	}
}


__global__ void cudaMatrixConv3D(float *data, float *kernel, float *Mout, int n_data, int p_data, int n_kernel, int p_kernel, int d_kernel){
	int row = threadIdx.x;
	int col = threadIdx.y;
	
	int p_C1 = p_data - p_kernel + 1;
	int n_C1 = n_data - n_kernel + 1;
	int index = row * p_C1 + col;
	
	int k = blockIdx.x; 
	
	int d_offset_kernel = k * (n_kernel * p_kernel);
	int d_offset_out = k * (n_C1 * p_C1);	
	
	float result = 0;
	for (int i = 0; i<n_kernel; i++){
		for(int j = 0; j<p_kernel; j++){
			result += data[(row + i) * p_data + (col + j)] * kernel[i * p_kernel + j + d_offset_kernel];
		}
	}
	
	Mout[index + d_offset_out]=result;
		
	
	
	
}

__global__ void cudaMatrixSubSampling3D(float *data, float *Mout, int n_data, int p_data, int d_data){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row>=0 && row<n_data/2 && col>=0 && col<p_data/2){
		for (int k = 0; k<d_data; k++){
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + row*2*n_data + col*2]/4.0;
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + row*2*n_data + col*2 + 1]/4.0;
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + (row*2+1)*n_data + col*2]/4.0;
			Mout[k*(n_data/2)*(p_data/2)+row*n_data/2 + col] += data[k*n_data*p_data + (row*2+1)*n_data + col*2+1]/4.0;
		}
	}
}


int main(void){
	srand(time(NULL));
	int n_raw_data = 32;
	int n_kernel = 5;
	int n_C1_data = n_raw_data - n_kernel +1;
	int n_S1_data = n_C1_data/2;
	int nb_kernel = 6;
	printf("n_C1_data=%d n_S1_data=%d \n", n_C1_data, n_S1_data);
	
	int raw_data_size = n_raw_data*n_raw_data*sizeof(float);
	int C1_data_size = nb_kernel*n_C1_data*n_C1_data*sizeof(float);
	int S1_data_size = nb_kernel*n_S1_data*n_S1_data*sizeof(float);
	int C1_kernel_size = nb_kernel*n_kernel*n_kernel*sizeof(float);
	
	
	float *raw_data = (float*) malloc(raw_data_size);
	float *C1_data = (float*) malloc(C1_data_size);
	float *S1_data = (float*) malloc(S1_data_size);
	float *C1_kernel = (float*) malloc(C1_kernel_size);
	
	MatrixInit(raw_data, n_raw_data, n_raw_data, 0, 1);
	MatrixInit(C1_data, nb_kernel, n_C1_data, n_C1_data, 0);
	MatrixInit(S1_data, nb_kernel, n_S1_data, n_S1_data, 0);
	MatrixInit(C1_kernel, nb_kernel, n_kernel, n_kernel, 1);
	
	float *raw_data_cu;
	float *C1_data_cu;
	float *S1_data_cu;
	float *C1_kernel_cu;
	
	hipMalloc((void **) &raw_data_cu, n_raw_data*n_raw_data*sizeof(float));
	hipMalloc((void **) &C1_data_cu, nb_kernel*n_C1_data*n_C1_data*sizeof(float));
	hipMalloc((void **) &S1_data_cu, nb_kernel*n_S1_data*n_S1_data*sizeof(float));
	hipMalloc((void **) &C1_kernel_cu, nb_kernel*n_kernel*n_kernel*sizeof(float));

	hipMemcpy(raw_data_cu, raw_data, n_raw_data*n_raw_data*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(C1_kernel_cu, C1_kernel, nb_kernel*n_kernel*n_kernel*sizeof(float), hipMemcpyHostToDevice);	
	
	dim3 dimGrid(nb_kernel,1,1);
	dim3 dimBlock(n_C1_data, n_C1_data);
	cudaMatrixConv3D<<<dimGrid, dimBlock>>>(raw_data_cu, C1_kernel_cu, C1_data_cu, n_raw_data, n_raw_data, n_kernel, n_kernel, nb_kernel);
	
	dim3 dimBlock_2(n_S1_data, n_S1_data);
	cudaMatrixSubSampling3D<<<1, dimBlock_2>>>(C1_data_cu, S1_data_cu, n_C1_data, n_C1_data, nb_kernel);
	
	hipMemcpy(S1_data, S1_data_cu, nb_kernel*n_S1_data*n_S1_data*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(C1_data, C1_data_cu, nb_kernel*n_C1_data*n_C1_data*sizeof(float), hipMemcpyDeviceToHost);
	
	
	printf("data : \n");
	MatrixPrint2D(raw_data, n_raw_data, n_raw_data);
	printf("kernels : \n");
	MatrixPrint3D(C1_kernel, nb_kernel, n_kernel, n_kernel);
	printf("conv result : \n");
	MatrixPrint3D(C1_data, nb_kernel, n_C1_data, n_C1_data);
	printf("sub samp result : \n");
	MatrixPrint3D(S1_data, nb_kernel, n_S1_data, n_S1_data);
	
}

	// TESTING
	
	//int n_kernel = 3;
	//int n_data = 6;
	//int d_kernel = 2;
	//float *test_data = (float*) malloc(n_data*n_data*sizeof(float));
	//float *test_kernel = (float*) malloc(n_kernel*n_kernel*d_kernel*sizeof(float));
	//MatrixInit(test_data, n_data, n_data, 0, 1);
	//MatrixInit(test_kernel, n_kernel, n_kernel, d_kernel, 1);
	
	
	//float *test_conv = (float*) malloc((n_data-n_kernel+1)*(n_data-n_kernel+1)*d_kernel*sizeof(float));
	//float *test_sub_samp = (float*) malloc(2*2*2*sizeof(float));
	
	//int N = n_data*n_data;
	//dim3 dimBlock(N,N);
	//dim3 dimGrid(ceil(N/16.0), ceil(N/16.0));
	//float *data_cu, *kernel_cu, *conv_cu, *sub_samp;
	//cudaMalloc((void **) &data_cu, n_data*n_data*sizeof(float));
	//cudaMalloc((void **) &kernel_cu, n_kernel*n_kernel*d_kernel*sizeof(float));
	//cudaMalloc((void **) &conv_cu, (n_data-n_kernel+1)*(n_data-n_kernel+1)*d_kernel*sizeof(float));
	//cudaMalloc((void **) &sub_samp, 2*2*2*sizeof(float));
	
	//cudaMemcpy(data_cu, test_data, n_data*n_data*sizeof(float), cudaMemcpyHostToDevice);
	//cudaMemcpy(kernel_cu, test_kernel, n_kernel*n_kernel*d_kernel*sizeof(float), cudaMemcpyHostToDevice);
	//cudaMatrixConv3D<<<dimBlock, dimGrid>>>(data_cu, kernel_cu, conv_cu, n_data, n_data, n_kernel, n_kernel, d_kernel);
	//cudaMatrixSubSampling3D<<<dimBlock, dimGrid>>>(conv_cu, sub_samp, 4, 4, 2);
	//cudaMemcpy(test_conv, conv_cu, (n_data-n_kernel+1)*(n_data-n_kernel+1)*d_kernel*sizeof(float), cudaMemcpyDeviceToHost);
	//cudaMemcpy(test_sub_samp, sub_samp, 2*2*2*sizeof(float), cudaMemcpyDeviceToHost);
